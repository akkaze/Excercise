#include "hip/hip_runtime.h"
#include "Test.h"

__global__ void vecAdd(float *a, float *b, float *c, int n) {
        int i = blockDim.x * blockIdx.x + threadIdx.x;

        if(i < n)
        {
        for(int j=0;j<1000000;++j)
                c[i] = a[i] / b[i];
				}
}

void Test::addVec_gpu(float *a, float *b, float *c, int n) {
        float *d_a, *d_b, *d_c;
        size_t size = n * sizeof(float);

        hipMalloc(&d_a, size);
        hipMalloc(&d_b, size);
        hipMalloc(&d_c, size);
				
				hipMemcpy(d_a, a, n*sizeof(float), hipMemcpyHostToDevice);
				hipMemcpy(d_b, b, n*sizeof(float), hipMemcpyHostToDevice);
				hipMemcpy(d_c, c, n*sizeof(float), hipMemcpyHostToDevice);
				vecAdd<<<(n/256)+1,256>>>(d_a,d_b,d_c,n);
				hipMemcpy(a, d_a, n*sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(b, d_b, n*sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(c, d_c, n*sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
}


