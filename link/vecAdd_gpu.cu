#include "hip/hip_runtime.h"
#include "Test.h"

void __global__  vecAdd(float *a, float *b, float *c, int n) {
        int i = blockDim.x * blockIdx.x + threadIdx.x;

        if(i < n)
        {
                c[i] = sqrt(a[i] * b[i]);
				}
}

void Test::addVec_gpu(float *a, float *b, float *c, int n) {
        float *d_a, *d_b, *d_c;
        size_t size = n * sizeof(float);

        hipMalloc(&d_a, size);
        hipMalloc(&d_b, size);
        hipMalloc(&d_c, size);
				
				hipMemcpy(d_a, a, n*sizeof(float), hipMemcpyHostToDevice);
				hipMemcpy(d_b, b, n*sizeof(float), hipMemcpyHostToDevice);
				hipMemcpy(d_c, c, n*sizeof(float), hipMemcpyHostToDevice);
				vecAdd<<<(n/16)+1,16>>>(d_a,d_b,d_c,n);
				hipMemcpy(a, d_a, n*sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(b, d_b, n*sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(c, d_c, n*sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
}


